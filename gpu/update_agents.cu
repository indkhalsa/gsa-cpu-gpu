#include "hip/hip_runtime.h"
/*
    Update velocity and position of each agent.

    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _UPDATE_AGENTS_
#define _UPDATE_AGENTS_

#include "gsa.h"

__global__ void update_agents( int n_dim, float *g, float *force, float *x_min, float *x_max, float *velocity, float *position, hiprandState *devState )
{
    int d, p=blockIdx.x;

    for(d=0; d<n_dim; d++)
    {
        velocity[d+p*n_dim] = ( hiprand_uniform(&devState[p]) * velocity[d+p*n_dim] ) + ( *g * force[d+p*n_dim] );

        if( ((position[d+p*n_dim] + velocity[d+p*n_dim]) < x_min[d]) || ((position[d+p*n_dim] + velocity[d+p*n_dim]) > x_max[d]) )
        {
            position[d+p*n_dim] = hiprand_uniform(&devState[p]) * ( x_max[d] - x_min[d] ) + x_min[d];
        }

        else
        {
            position[d+p*n_dim] = position[d+p*n_dim] + velocity[d+p*n_dim];
        }
    }
}

#endif
