#include "hip/hip_runtime.h"
/*
    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "gsa.h"

int main( int argc, char **argv )
{
    // Host variables
    int d, p;
    int n_pop=5, n_dim=2, i=0, max_i=100, func=1;
    float min=0.0, max=0.0, alpha=20, g0=100, epsilon=0.1;
    float *x, *z;
    struct timeval tvBegin, tvEnd, tvDiff;

    // Device variables
    hiprandState *devState;
    float *sum, *g, *best, *worst;  
    float *position, *velocity, *acceleration, *x_min, *x_max, *fitness, *mass, *inertia_mass, *force;

    // Initialize
    init_params( argc, argv, &n_pop, &n_dim, &min, &max, &max_i, &alpha, &g0, &epsilon, &func );

    hipMalloc( (void **)&devState, n_pop * sizeof(hiprandState) );

    hipMalloc( (void **)&sum, sizeof(float) );
    hipMalloc( (void **)&g, sizeof(float) );
    hipMalloc( (void **)&best, sizeof(float) );
    hipMalloc( (void **)&worst, sizeof(float) );

    hipMalloc( (void **)&position, n_pop * n_dim * sizeof(float) );
    hipMalloc( (void **)&velocity, n_pop * n_dim * sizeof(float) );
    hipMalloc( (void **)&acceleration, n_pop * n_dim * sizeof(float) );
    hipMalloc( (void **)&x_min, n_dim * sizeof(float) );
    hipMalloc( (void **)&x_max, n_dim * sizeof(float) );
    hipMalloc( (void **)&fitness, n_pop * sizeof(float) );
    hipMalloc( (void **)&mass, n_pop * sizeof(float) );
    hipMalloc( (void **)&inertia_mass, n_pop * sizeof(float) );
    hipMalloc( (void **)&force, n_pop * n_dim * sizeof(float) );

    x = (float *)calloc( n_pop * n_dim, sizeof(float) );
    z = (float *)calloc( n_pop, sizeof(float) );

    gettimeofday(&tvBegin, NULL);

    init_gsa<<<n_pop, 1>>>( n_dim, min, max, x_min, x_max, position, velocity, devState, (unsigned int)time(NULL) );

    // Start
    while( i<max_i )
    {
        get_fitness<<<n_pop, 1>>>( func, n_dim, position, fitness );
        get_gconst<<<1, 1>>>( i, max_i, alpha, g0, g );
        get_best_worst<<<1, 1>>>( n_pop, fitness, best, worst );
        get_mass<<<n_pop, 1>>>( best, worst, fitness, mass );
        get_sum_masses<<<1, 1>>>( n_pop, mass, sum );
        get_inertia_mass<<<n_pop, 1>>>( mass, sum, inertia_mass );
        get_forces<<<n_pop, 1>>>( n_pop, n_dim, epsilon, position, inertia_mass, force, devState );
        update_agents<<<n_pop, 1>>>( n_dim, g, force, x_min, x_max, velocity, position, devState );

        i++;
    }

    gettimeofday(&tvEnd, NULL);
    timeval_subtract(&tvDiff, &tvEnd, &tvBegin);

    hipMemcpy( x, position, n_pop * n_dim * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( z, fitness, n_pop * sizeof(float), hipMemcpyDeviceToHost );

    // Get best solution
    p = get_best_solution( n_pop, z );

    // Show solution  
    printf( "\nAfter %d iterations...\n\n", max_i );

    for( d=0; d<n_dim; d++ )
    {
        printf( "x(%d): %f\n", d+1, x[d+p*n_dim] );
    }

    printf( "\nz: %f\n\n", z[p] );

    printf("t: %ld.%06ld s\n\n", tvDiff.tv_sec, tvDiff.tv_usec);

    // Free memory  
    hipFree( devState ); hipFree( sum ); hipFree( g ); hipFree( best ); hipFree( worst );
    hipFree( position ); hipFree( velocity ); hipFree( acceleration ); hipFree( x_min ); hipFree( x_max );
    hipFree( fitness ); hipFree( mass ); hipFree( inertia_mass ); hipFree( force ); free( x ); free( z );

    return 0;
}
