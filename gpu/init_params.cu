#include "hip/hip_runtime.h"
/*
    Parameters initialization.

    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _INIT_PARAMS_
#define _INIT_PARAMS_

#include "gsa.h"

void init_params( int argc, char **argv, int *n_pop, int *n_dim, float *min, float *max, int *max_i, float *alpha, float *g0, float *epsilon, int *func )
{
    char c;

    while( (c = getopt(argc, argv, "a:d:x:X:i:A:G:e:f:h")) != -1 )
    {
        switch( c )
        {
            case 'a':
                *n_pop = atoi(optarg);
	            break;
            case 'd':
                *n_dim = atoi(optarg);
                break;
            case 'x':
                *min = atof(optarg);
                break;
            case 'X':
	            *max = atof(optarg);
                break;
            case 'i':
                *max_i = atoi(optarg);
                break;
            case 'A':
                *alpha = atof(optarg);
                break;
            case 'G':
                *g0 = atof(optarg);
                break;
            case 'e':
                *epsilon = atof(optarg);
                break;
            case 'f':
                *func = atoi(optarg);
                break;
            case 'h':
                usage();
                break;
            default:
                usage();
                break;
        }
    }

    if( argc < 8 )
    {
        exit(EXIT_FAILURE);
    }
}

#endif
