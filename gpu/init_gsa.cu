#include "hip/hip_runtime.h"
/*
    Gravitational search algorithm initialization.

    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _INIT_GSA_
#define _INIT_GSA_

#include "gsa.h"

__global__ void init_gsa( int n_dim, float min, float max, float *x_min, float *x_max, float *position, float *velocity, hiprandState *devState, unsigned int seed )
{
    int d, p=blockIdx.x;

    hiprand_init( seed, p, 0, &devState[p] );

    if( p == 0 )
    {
        // Boundaries
        for( d=0; d<n_dim; d++ )
        {
            x_min[d] = min;
            x_max[d] = max;
        }
    }

    for( d=0; d<n_dim; d++ )
    {
        // Position
        position[d+p*n_dim] = hiprand_uniform(&devState[p]) * (x_max[d] - x_min[d]) + x_min[d];

        // Velocity
        velocity[d+p*n_dim] = 0.0f;
    }
}

#endif
