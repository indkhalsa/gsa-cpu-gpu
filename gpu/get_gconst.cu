#include "hip/hip_runtime.h"
/*
    G Constant.

    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _GET_GCONST_
#define _GET_GCONST_

#include "gsa.h"

__global__ void get_gconst( int i, int max_i, float alpha, float g0, float *g )
{
    *g = ( g0 * exp(-alpha * i/max_i) );
}

#endif
