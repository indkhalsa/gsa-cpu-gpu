#include "hip/hip_runtime.h"
/*
    Get the total force in different directions.

    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _GET_FORCES_
#define _GET_FORCES_

#include "gsa.h"

__global__ void get_forces( int n_pop, int n_dim, float epsilon, float *position, float *inertia_mass, float *force, hiprandState *devState )
{
    int d, l, k, p=blockIdx.x;
    float r, sum;

    for( d=0; d<n_dim; d++ )
    {
        force[d+p*n_dim] = 0.0f;

        for( k=0; k<n_pop; k++ )
        {
            if( k != p )
            {
                sum = 0.0f;

                for( l=0; l<n_dim; l++ )
                {
                    sum = sum + powf( (position[l+k*n_dim] - position[l+p*n_dim]), 2 );
                }

                r = powf( sum, 0.5 );

                force[d+p*n_dim] = force[d+p*n_dim] + ( hiprand_uniform(&devState[p]) * inertia_mass[k] * (position[d+k*n_dim] - position[d+p*n_dim]) / (r + epsilon) );
            }
        }
    }
}

#endif
