#include "hip/hip_runtime.h"
/*
    Get the best and worst fitness value.

    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _GET_BEST_WORST_
#define _GET_BEST_WORST_

#include "gsa.h"

__global__ void get_best_worst( int n_pop, float *fitness, float *best, float *worst )
{
    int p;
    float fmin = fitness[0], fmax = fitness[0];

    for( p=1; p<n_pop; p++ )
    {
        if( fitness[p] < fmin )
            fmin = fitness[p];

        if( fitness[p] > fmax )
            fmax = fitness[p];
    }

  *best = fmin; *worst = fmax;
}

#endif
