#include "hip/hip_runtime.h"
/*
    Get the fitness value of each agent according to their position.

    Copyright (C) 2011  Edgard Nikitiuk <edgnik@gmail.com>
 
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
 
    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
 
    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _GET_FITNESS_
#define _GET_FITNESS_

#include "gsa.h"

__global__ void get_fitness( int func, int n_dim, float *position, float *fitness )
{
    int i, j, d, m=10, p=blockIdx.x;
    float pi = 3.141592654f, a=20.0f, b=0.2f, c=2*pi;
    float sum=0.0f, sum_i=0.0f, sum_j=0.0f, sum_pow=0.0f, sum_cos=0.0f, prod_cos=0.0f;

    switch( func )
    {
        // De Jong
        case 1:
            for( d=0; d<n_dim; d++ )
            {
                sum = sum + powf(position[d+p*n_dim], 2);
            }

            fitness[p] = sum;
            break;

        // Sphere
        case 2:
            for( d=0; d<n_dim; d++ )
            {
                sum = sum + ( (d+1) * powf(position[d+p*n_dim], 2) );
            }

            fitness[p] = sum;
            break;

        // Rotated hyper-ellipsoid
        case 3:
            for( i=0; i<n_dim; i++ )
            {
                sum_j = 0.0;

                for( j=0; j<(i+1); j++ )
                {
                    sum_j = sum_j + powf(position[j+p*n_dim], 2);
                }

                sum_i = sum_i + sum_j;
            }

            fitness[p] = sum_i;
            break;

        // Rosenbrock
        case 4:
            for( d=0; d<(n_dim-1); d++ )
            {
                sum = sum + ( powf((1.0f - position[d+p*n_dim]), 2) + (100.0f * powf((position[d+p*n_dim+1] - powf(position[d+p*n_dim], 2)), 2)) );
            }

            fitness[p] = sum;
            break;

        // Rastrigin
        case 5:
            for( d=0; d<n_dim; d++ )
            {
                sum = sum + ( powf(position[d+p*n_dim], 2) - (10.0f * cos(2 * pi * position[d+p*n_dim])) );
            }

            fitness[p] = (10.0f * n_dim) + sum;
            break;

        // Schwefel
        case 6:
            for( d=0; d<n_dim; d++ )
            {
                sum = sum + ( -position[d+p*n_dim] * sin(powf(fabs(position[d+p*n_dim]), 0.5)) );
            }

            fitness[p] = sum;
            break;

        // Griewangk
        case 7:
            sum_pow = pow(position[p*n_dim], 2); prod_cos = cos(position[p*n_dim]/sqrtf(1));

            for( d=1; d<n_dim; d++ )
            {
                sum_pow = sum_pow + powf(position[d+p*n_dim], 2);
                prod_cos = prod_cos * cos(position[d+p*n_dim]/sqrtf(d+1));
            }

            fitness[p] = (sum_pow/4000) - prod_cos + 1;
            break;

        // Ackley
        case 8:
            for( d=0; d<n_dim; d++ )
            {
                sum_pow = sum_pow + powf(position[d+p*n_dim], 2);
                sum_cos = sum_cos + cos(c*position[d+p*n_dim]);
            }

            fitness[p] = -a * expf(-b * sqrtf(sum_pow/n_dim) - expf(sum_cos/n_dim)) + a + expf(1);
            break;

        // Michalewicz
        case 9:
            for( d=0; d<n_dim; d++ )
            {
                sum = sum + ( sin(position[d+p*n_dim]) * powf(sin((d+1) * powf(position[d+p*n_dim], 2) / pi), 2*m) );
            }

            fitness[p] = -sum;
            break;
    }
}

#endif
